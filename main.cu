#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "inline_check.h"

template<typename EQ>
__global__ void testKernel(int N) {

    // N3928 - static assert without message
    static_assert(true);

    // P0217R3 - structured binding
    int a[2] = {1, 2};
    auto [a1, a2] = a;
    assert(a1 == 1);
    assert(a2 == 2);

    // P0386R2 - inline variables
    assert(EQ::M == 2);

    // P0292R2 - constexpr if statements
    if constexpr(EQ::M ==2) {
        printf("M == 2\n");
    } else {
        printf("M != 2\n");
    }

    int gtid = blockIdx.x*blockDim.x + threadIdx.x;
    assert(gtid < N);
}

int main(int argc, char **argv) {
    printf("starting...\n");
    testKernel<StaticMembers><<<256, 256>>>(60);
    exit(0);
}
